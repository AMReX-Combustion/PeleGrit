
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime_api.h>

__global__ void parallel_for(const int n, double* da) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if (tid < n) {
        double dummy = 123.456;
        da[tid] = dummy + 123.456*dummy;
    }
}

int main()
{
  const int N = 10000000;
  int blockSize = 256;
  int numBlocks = (N + blockSize -1) / blockSize;

  double* da;

  hipMalloc((void**)&da, sizeof(double)*N);

  //warm up
  for(int j=0; j<10; j++)
  {
    parallel_for<<<numBlocks, blockSize>>>(N, da);
  }

  typedef std::chrono::high_resolution_clock Time;
  typedef std::chrono::duration<float> fsec;

  hipDeviceSynchronize();
  auto start_clock = Time::now();

  hipProfilerStart();

  for(int j=0; j<10; j++)
  {
    parallel_for<<<numBlocks, blockSize>>>(N, da);
  }
  
  hipDeviceSynchronize();
  
  hipProfilerStop();

  auto finish_clock = Time::now();
  fsec fs = finish_clock - start_clock;
  std::cout << "time taken for cuda parallel for (msecs):" << fs.count()*1e3 << std::endl;

  hipFree(da);

  return 0; 
}
